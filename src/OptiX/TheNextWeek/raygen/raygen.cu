#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "raydata.cuh"
#include "random.cuh"
#include "camera.cuh"

using namespace optix;

// Optix program built-in indices
rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, theLaunchDim, rtLaunchDim, );

// Ray state variables
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(PerRayData, thePrd, rtPayload,  );

// "Global" variables
rtDeclareVariable(rtObject, sysWorld, , );
rtBuffer<float3, 2> sysOutputBuffer;

// Ray Generation variables
rtDeclareVariable(int, numSamples, , );
rtDeclareVariable(int, maxRayDepth, , );

inline __device__ float3 removeNaNs(float3 radiance)
{
    float3 r = radiance;
    if(!(r.x == r.x)) r.x = 0.0f;
    if(!(r.y == r.y)) r.y = 0.0f;
    if(!(r.z == r.z)) r.z = 0.0f;
    return r;
}

inline __device__ float3 missColor(const optix::Ray &theRay)
{
  float3 unitDirection = normalize(theRay.direction);
  float t = 0.5f * (unitDirection.y + 1.0f);
  // "sky" gradient
  float3 missColor = (1.0f-t) * make_float3(1.0f, 1.0f, 1.0f)
      + t * make_float3(0.5f, 0.7f, 1.0f);

  return missColor;
}


inline __device__ float3 color(optix::Ray& theRay, uint32_t& seed)
{
    PerRayData thePrd;
    thePrd.seed = seed;
    float3 sampleRadiance = make_float3(1.0f, 1.0f, 1.0f);
    thePrd.gatherTime = cameraTime0 + randf(seed)*(cameraTime1 - cameraTime0);

    for(int i = 0; i < maxRayDepth; i++)
    {
        rtTrace(sysWorld, theRay, thePrd);
        if (thePrd.scatterEvent == Ray_Miss)
        {
            return sampleRadiance * missColor(theRay);
        }
        else if (thePrd.scatterEvent == Ray_Finish)
        {
            return sampleRadiance * thePrd.attenuation;
        }
        else if (thePrd.scatterEvent == Ray_Cancel)
        {
            return make_float3(0.f);
        }
        else
        {
            // ray is still alive, and got properly bounced
            sampleRadiance *= thePrd.attenuation;
            theRay = optix::make_Ray(
                thePrd.scattered_origin,
                thePrd.scattered_direction,
                0,
                1e-3f,
                RT_DEFAULT_MAX
            );
        }
    }

    seed = thePrd.seed;

    return make_float3(0.0f);
}

RT_PROGRAM void rayGenProgram()
{
    uint32_t seed = tea<64>(theLaunchDim.x * theLaunchIndex.y + theLaunchIndex.x, 0);

    float3 radiance = make_float3(0.0f, 0.0f, 0.0f);
    for (int n = 0; n < numSamples; n++)
    {
        float s = float(theLaunchIndex.x+randf(seed)) / float(theLaunchDim.x);
        float t = float(theLaunchIndex.y+randf(seed)) / float(theLaunchDim.y);
        optix::Ray theRay = generateRay(s,t);
        float3 sampleRadiance = color(theRay, seed);

        // Remove NaNs
        //sampleRadiance = removeNaNs(sampleRadiance);

        radiance += sampleRadiance;
    }
    radiance /= numSamples;

    // gamma correction (2)
    radiance = make_float3(
        sqrtf(radiance.x),
        sqrtf(radiance.y),
        sqrtf(radiance.z)
    );

    sysOutputBuffer[theLaunchIndex] = radiance;
}
