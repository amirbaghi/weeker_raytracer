#include "hip/hip_runtime.h"
#include <optix.h>

#include "raydata.cuh"
#include "sampling.cuh"

// Ray state variables
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(PerRayData, thePrd, rtPayload,  );

// "Global" variables
rtDeclareVariable(rtObject, sysWorld, , );

// The point and normal of intersection
rtDeclareVariable(HitRecord, hitRecord, attribute hitRecord, );

// Material variables
rtDeclareVariable(float3, color, , );

RT_PROGRAM void closestHit()
{
    float3 scatterDirection = hitRecord.normal + randomInUnitSphere(thePrd.seed);

    thePrd.scatterEvent = Ray_Hit;
    thePrd.scattered_origin = hitRecord.point;
    thePrd.scattered_direction = scatterDirection;
    thePrd.attenuation = color;
}
