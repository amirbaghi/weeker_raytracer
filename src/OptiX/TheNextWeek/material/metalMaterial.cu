#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "raydata.cuh"
#include "sampling.cuh"

// Ray state variables
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(PerRayData, thePrd, rtPayload,  );

// "Global" variables
rtDeclareVariable(rtObject, sysWorld, , );

// The point and normal of intersection
rtDeclareVariable(HitRecord, hitRecord, attribute hitRecord, );

// Material variables
rtDeclareVariable(float3, color, , );
rtDeclareVariable(float, roughness, , );

rtDeclareVariable(rtCallableProgramId<float3(float, float, float3)>, sampleTexture, , );


RT_PROGRAM void closestHit()
{
    float3 scatterDirection = optix::reflect(theRay.direction, hitRecord.normal) +
        roughness*randomInUnitSphere(thePrd.seed);

    if (optix::dot(scatterDirection, hitRecord.normal) <= 0.0f )
    { // Ray is absorbed by the material
        thePrd.scatterEvent = Ray_Finish;
        // thePrd.scattered_origin = hitRecord.point;
        // thePrd.scattered_direction = scatterDirection;
        thePrd.attenuation = make_float3(0.0f, 0.0f, 0.0f);
        return;
    }

    thePrd.scatterEvent = Ray_Hit;
    thePrd.scattered_origin = hitRecord.point;
    thePrd.scattered_direction = scatterDirection;
    thePrd.attenuation = color;
    //thePrd.attenuation = sampleTexture(0.f, 0.f, hitRecord.point);
}
